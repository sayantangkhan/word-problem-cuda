#include "fsa_reader.c"
#include "multiplier.cu"
#include "misc.cu"
#include <stdio.h>

typedef struct Word {
  int length;
  int* word;
} Word;

Word parse_word(char* filename) {
  FILE* fp = fopen(filename, "r");
  Word word;
  fscanf(fp, "%d", &word.length);
  word.word = (int *) malloc(sizeof(int) * word.length);
  int i;
  for (i=0; i<word.length; i++) {
    fscanf(fp, "%d", &word.word[i]);
  }
  return word;
}

typedef struct WordList {
  int length;
  Word* words;
} WordList;

WordList parse_multiple_words(char* filename) {
  WordList word_list;
  FILE* fp = fopen(filename, "r");
  fscanf(fp, "%d", &word_list.length);
  word_list.words = (Word *) malloc(sizeof(Word) * word_list.length);
  int j;
  for (j=0; j<word_list.length; j++) {
    Word word;
    fscanf(fp, "%d", &word.length);
    word.word = (int *) malloc(sizeof(int) * word.length);
    int i;
    for (i=0; i<word.length; i++) {
      fscanf(fp, "%d", &word.word[i]);
    }
    word_list.words[j] = word;
  }
  return word_list;
}

int main(int argc, char* argv[]) {
  if (argc < 3) {
    exit(-1);
  }

  char* hg_filename = argv[1];
  char* wordlist_filename = argv[2];

  HyperbolicGroup hyperbolic_group = parse_hyperbolic_group(hg_filename);
  host_hyperbolic_group = (HyperbolicGroup *) malloc(sizeof(HyperbolicGroup));
  memcpy(host_hyperbolic_group, &hyperbolic_group, sizeof(HyperbolicGroup));

  WordAcceptor word_acceptor = host_hyperbolic_group->word_acceptor;
  GeneralMultiplier general_multiplier = host_hyperbolic_group->general_multiplier;
  WordList word_list = parse_multiple_words(wordlist_filename);

  copy_hg_to_device();

  int j;
  for (j=0; j<word_list.length; j++) {
    Word word = word_list.words[j];

    int generator_to_multiply = 1;
    int* result = (int*) malloc(sizeof(int) * (word.length + 1));

    // Starting recording
    float elapsed_milliseconds = 0;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Actual computation
    multiply_with_generator(word.length, word.word, generator_to_multiply, result);
    hipDeviceSynchronize();

    // Stopping recording
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_milliseconds, start, stop);

    // Printing elapsed time
    printf("%d, %f\n", word.length, elapsed_milliseconds);
  }

  exit(0);
}

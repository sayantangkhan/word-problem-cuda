#include "hip/hip_runtime.h"
#include "fsa_reader.c"
#include <stdio.h>

#define BLOCK_SIZE 64 // Tweak later

typedef struct Slice {
  int start_index; // Inclusive
  int end_index; // Exclusive
} Slice;

__host__ void copy_general_multiplier(GeneralMultiplier* host_general_multiplier, GeneralMultiplier* device_general_multiplier) {
  hipMemcpy(&device_general_multiplier->alphabet_size, &host_general_multiplier->alphabet_size, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(&device_general_multiplier->num_states, &host_general_multiplier->num_states, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(&device_general_multiplier->initial_state, &host_general_multiplier->initial_state, sizeof(int), hipMemcpyHostToDevice);

  int binary_alphabet_size = host_general_multiplier->alphabet_size * host_general_multiplier->alphabet_size;

  int *device_accepting_states, *device_transition_matrix;
  hipMalloc(&device_accepting_states, sizeof(int) * host_general_multiplier->num_states);
  hipMalloc(&device_transition_matrix, sizeof(int) * host_general_multiplier->num_states * binary_alphabet_size);
  hipMemcpy(device_accepting_states, host_general_multiplier->accepting_states, sizeof(int) * host_general_multiplier->num_states, hipMemcpyHostToDevice);
  hipMemcpy(device_transition_matrix, host_general_multiplier->transition_matrix, sizeof(int) * host_general_multiplier->num_states * binary_alphabet_size, hipMemcpyHostToDevice);

  hipMemcpy(&device_general_multiplier->accepting_states, &device_accepting_states, sizeof(int*), hipMemcpyHostToDevice);
  hipMemcpy(&device_general_multiplier->transition_matrix, &device_transition_matrix, sizeof(int*), hipMemcpyHostToDevice);
}

// __global__ void diagnostics(GeneralMultiplier* general_multiplier) {
//   printf("General multiplier alphabet = %d\n", general_multiplier->alphabet_size);
//   printf("General multiplier states = %d\n", general_multiplier->num_states);
//   printf("General multiplier initial state = %d\n", general_multiplier->initial_state);
//   int initial_state = 1;
//   int letter = 8;
//   /* int width = word_acceptor.alphabet_size; */
//   int width = (general_multiplier->alphabet_size) * (general_multiplier->alphabet_size);
//   printf("%d\n", general_multiplier->transition_matrix[initial_state * width + letter]);
// }

// __global__ void size_one_diagnostic(int* internal_path_matrix) {
//   printf("SOD starting\n");
//   int initial_state = 1;
//   int final_state = 5;
//   int word_index = 2;
//   int word_length = 11;
//   int path = internal_path_matrix[(initial_state * 105 + final_state)*word_length + word_index];
//   printf("SOD %d\n", path);
// }

__global__ void populate_slices(int word_length, GeneralMultiplier* general_multiplier, Slice* slices) {
  int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_thread_id < word_length) {
      Slice slice = {global_thread_id, global_thread_id + 1};
      slices[global_thread_id] = slice;
    }
}

__global__ void compute_size_one_paths(int word_length, int* word, GeneralMultiplier* general_multiplier, Slice* slices, int* internal_path_matrix) {
  int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int num_states = general_multiplier->num_states;
  if (global_thread_id < word_length * num_states) {
    int initial_state = global_thread_id / word_length;
    int word_index = global_thread_id % word_length;
    int alphabet_size = general_multiplier->alphabet_size;
    int i = word[word_index];
    int j;
    for (j=0; j<alphabet_size; j++) {
      int transition_matrix_index = initial_state * (alphabet_size * alphabet_size) + (i*alphabet_size + j);
      int final_state = general_multiplier->transition_matrix[transition_matrix_index];
      int internal_path_matrix_index = ((initial_state * num_states) + final_state) * word_length + word_index;
      internal_path_matrix[internal_path_matrix_index] = j;
    }
  }
}

int multiply_with_generator(int word_length, int* word, int generator_to_multiply, GeneralMultiplier* device_general_multiplier, GeneralMultiplier* host_general_multiplier, int* result) {
  int* device_word;
  Slice *slices;
  int *internal_path_matrix, *temp_buffer;
  int num_states = host_general_multiplier->num_states;

  int padding_symbol = host_general_multiplier->alphabet_size - 1;
  hipMalloc(&device_word, sizeof(int) * (word_length + 1));
  hipMemcpy(device_word, word, sizeof(int) * word_length, hipMemcpyHostToDevice);
  hipMemcpy(&device_word[word_length], &padding_symbol, sizeof(int), hipMemcpyHostToDevice); // Adding the padding symbol

  // Allocating memory for path matrices
  hipMalloc(&slices, sizeof(Slice) * (word_length + 1));
  hipMalloc(&internal_path_matrix, sizeof(int) * num_states * num_states * (word_length + 1));
  hipMemset(internal_path_matrix, -1, sizeof(int) * num_states * num_states * (word_length + 1));
  hipMalloc(&temp_buffer, sizeof(int) * num_states * num_states * (word_length + 1));
  hipMemset(temp_buffer, -1, sizeof(int) * num_states * num_states * (word_length + 1));

  int num_blocks;
  if ((word_length + 1) % BLOCK_SIZE == 0) {
    num_blocks = (word_length + 1)/BLOCK_SIZE;
  } else {
    num_blocks = (word_length + 1)/BLOCK_SIZE + 1;
  }
  populate_slices<<<num_blocks, BLOCK_SIZE>>>(word_length + 1, device_general_multiplier, slices);

  if (((word_length + 1)*num_states) % BLOCK_SIZE == 0) {
    num_blocks = ((word_length + 1) * num_states)/BLOCK_SIZE;
  } else {
    num_blocks = ((word_length + 1) * num_states)/BLOCK_SIZE + 1;
  }
  compute_size_one_paths<<<num_blocks, BLOCK_SIZE>>>(word_length + 1, device_word, device_general_multiplier, slices, internal_path_matrix);

  // size_one_diagnostic<<<1,1>>>(internal_path_matrix);
  // hipDeviceSynchronize();

  return 0; // Temporary
}
